#include "hip/hip_runtime.h"
#include <iostream>
#include <torch/torch.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <chrono>
#include <ATen/Dispatch.h>

// #include <THC/THCAtomics.cuh>
// #include <THC/THCGeneral.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <ATen/ATen.h>
// #include <ATen/AccumulateType.h>
// #include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
// #include <ATen/cuda/CUDAGraphsUtils.cuh>
// #include <c10/macros/Macros.h>
// #include <ATen/native/TensorIterator.h>
// #include <ATen/native/cuda/Loops.cuh>
using torch::IntArrayRef;
using torch::Tensor;
#define RN_NUM_THREADS 512
__global__ void test_rand_kernel(
    at::cuda::detail::TensorInfo<float, int64_t> output_info,
    std::pair<uint64_t, uint64_t> seeds,
    int N
){
    const int64_t id = blockIdx.x*blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seeds.first, id, seeds.second, &state);
    float rand = hiprand_uniform(&state);
    if(id < N){
        const int64_t offset = at::cuda::detail::IndexToOffset<float, int64_t, 1>::get(id, output_info);
        output_info.data[offset] = rand;
    }

}

Tensor test_rand_cuda(int N){
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, 0);
    Tensor output = torch::ones(N, options);
    int64_t block_size = RN_NUM_THREADS;
    dim3 dim_block(block_size);
    dim3 dim_grid((N + block_size - 1) / block_size);
    auto gen = at::check_generator<at::CUDAGeneratorImpl>(at::cuda::detail::getDefaultCUDAGenerator());
    uint64_t seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::cout<<"seed:"<<seed<<std::endl;
    // seed = 1690117527861138371;
    gen->set_current_seed(1);
    std::pair<uint64_t, uint64_t> rng_engine_inputs;
    {
        // See Note [Acquire lock when using random generators]
        std::lock_guard<std::mutex> lock(gen->mutex_);
        rng_engine_inputs = gen->philox_engine_inputs(N);
    }
    std::cout<<rng_engine_inputs.first<<" "<<rng_engine_inputs.second<<std::endl;
    auto output_info = at::cuda::detail::getTensorInfo<float, int64_t>(output);

    test_rand_kernel<<<dim_grid, dim_block>>>(output_info, rng_engine_inputs, N);
    return output;
}
#define LOW_MEM_DROPOUT_NUM_THREADS 512
template <typename scalar_t, int ADims, int BDims = ADims>
__global__ void low_mem_dropout_forward_kernel(
    at::cuda::detail::TensorInfo<scalar_t, int64_t> data_info,
    at::cuda::detail::TensorInfo<scalar_t, int64_t> output_info,
    at::cuda::detail::TensorInfo<bool, int64_t> mask_info,
    // at::cuda::detail::TensorInfo<float, int64_t> rand_num_info,
    std::pair<uint64_t, uint64_t> seeds,
    uint64_t N,
    float p
){
    const uint64_t id = blockIdx.x*blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seeds.first, id, seeds.second, &state);
    float rand = hiprand_uniform(&state);
    if(id < N){
        const int64_t data_offset = at::cuda::detail::IndexToOffset<scalar_t, int64_t, 1>::get(id, data_info);
        const int64_t output_offset = at::cuda::detail::IndexToOffset<scalar_t, int64_t, 1>::get(id, output_info);
        const int64_t mask_offset = at::cuda::detail::IndexToOffset<bool, int64_t, 1>::get(id, mask_info);
        // const int64_t rand_num_offset = at::cuda::detail::IndexToOffset<float, int64_t, 1>::get(id, rand_num_info);
        // rand_num_info.data[rand_num_offset] = rand;
        mask_info.data[mask_offset] = rand>p;
        scalar_t data = data_info.data[data_offset];
        scalar_t output = data*(rand>p);
        output_info.data[output_offset] = output;
    }
}

std::pair<Tensor, Tensor> low_mem_dropout_forward_cuda(Tensor data, float p){
    uint64_t n_elements = 1;
    for (size_t i = 0; i < data.dim(); ++i) {
        n_elements *= data.size(i);
    }
    std::cout<<"n_elements:"<<n_elements<<std::endl;
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(data.device());
    Tensor output = torch::empty_like(data, options);
    options = torch::TensorOptions().dtype(torch::kBool).device(data.device());
    Tensor mask = torch::zeros_like(data, options);
    uint64_t block_size = LOW_MEM_DROPOUT_NUM_THREADS;
    unsigned int blocks_per_sm = at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor/block_size;
    // unsigned int n_blocks = at::cuda::getCurrentDeviceProperties()->multiProcessorCount * blocks_per_sm;
    dim3 dim_block(block_size);
    dim3 grid((n_elements + block_size - 1) / block_size);
    // grid.x = std::min((unsigned int)at::cuda::getCurrentDeviceProperties()->multiProcessorCount * blocks_per_sm, grid.x);
    // uint64_t counter_offset = ((n_elements - 1)/(block_size*grid.x)+1);
    auto gen = at::check_generator<at::CUDAGeneratorImpl>(at::cuda::detail::getDefaultCUDAGenerator());
    uint64_t seed = std::chrono::system_clock::now().time_since_epoch().count();
    gen->set_current_seed(seed);
    std::pair<uint64_t, uint64_t> rng_engine_inputs;
    {
        // See Note [Acquire lock when using random generators]
        std::lock_guard<std::mutex> lock(gen->mutex_);
        rng_engine_inputs = gen->philox_engine_inputs(n_elements);
    }
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "low_mem_dropout_forward", ([&] {
        auto data_info = at::cuda::detail::getTensorInfo<scalar_t, int64_t>(data);
        auto output_info = at::cuda::detail::getTensorInfo<scalar_t, int64_t>(output);
        auto mask_info = at::cuda::detail::getTensorInfo<bool, int64_t>(mask);
        data_info.collapseDims();
        output_info.collapseDims();
        mask_info.collapseDims();
        low_mem_dropout_forward_kernel<scalar_t, 1><<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
            data_info, output_info, mask_info, //rand_num_info,
            rng_engine_inputs,
            n_elements, p);
    }));
    return std::make_pair(output, mask);

}